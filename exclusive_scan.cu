
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<math.h>

#define N 512

__global__ void exclusive_scan(int *d_in) {

	__shared__ int temp_in[N];

	int id = threadIdx.x;
	temp_in[id] = d_in[id];

	__syncthreads();

	unsigned int s = 1;
	//int i = 2*s*(threadIdx.x + 1) - 1;
	for(; s <= N-1; s <<= 1) {
		int i = 2*s*(threadIdx.x + 1) - 1;
		if((i >= s) && (i < N)) {
			int a = temp_in[i];
			int b = temp_in[i-s];
			int c = a + b;
			temp_in[i] = c;
		}
		__syncthreads();
	}

	//d_in[i] = temp_in[i];

	if(threadIdx.x == 0) {
		d_in[N-1] = 0;
		temp_in[N-1] = 0;
	}

	for(s = s/2; s >= 1; s >>= 1) {
		int i = 2*s*(threadIdx.x + 1) - 1;
		if((i <= s) && (i < N)) {
			int r = temp_in[i];
			int l = temp_in[i-s];
			__syncthreads();
			temp_in[i] = l + r;
			temp_in[i-s] = r;
		}
		__syncthreads();
	}

	d_in[id] = temp_in[id];

}

int main()
{
	int h_in[N];
	int h_out[N];

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	for(int i=0; i < N; i++)
		h_in[i] = 1;

	int *d_in;
	//int *d_out;

	hipMalloc((void**) &d_in, N*sizeof(int));
	//cudaMalloc((void**) &d_out, N*sizeof(int));
	hipMemcpy(d_in, &h_in, N*sizeof(int), hipMemcpyHostToDevice);
	
	hipEventRecord(start);

	//Implementing kernel call
	exclusive_scan<<<1, N>>>(d_in);

	hipEventRecord(stop);
	hipEventSynchronize(stop);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(&h_out, d_in, N*sizeof(int), hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++)
		printf("out[%d] = %d\n", i, h_out[i]);

	hipFree(d_in);
	//cudaFree(d_out);

	printf("Time used: %f milliseconds\n", milliseconds);

	return -1;

}
